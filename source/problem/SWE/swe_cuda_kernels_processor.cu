#include "hip/hip_runtime.h"
#include "swe_cuda_kernels_processor.hpp"
#include "swe_definitions.hpp"
#include "swe_true_src_functions.hpp"

template <typename ElementType>
__global__ void cuda_volume_kernel1(ElementType& elt, uint n_gp) {
    auto& internal = elt.data.internal;
    uint gp = blockDim.x * blockIdx.x + threadIdx.x;
    if (gp < n_gp) {
        internal.water_column_hgt_at_gp[gp] = internal.ze_at_gp[gp] + internal.bath_at_gp[gp];
        internal.ze_flux_at_gp[GlobalCoord::x][gp] = internal.qx_at_gp[gp];
        internal.ze_flux_at_gp[GlobalCoord::y][gp] = internal.qy_at_gp[gp];
        internal.qx_flux_at_gp[GlobalCoord::x][gp] = pow(internal.qx_at_gp[gp], 2) /
                internal.water_column_hgt_at_gp[gp] +
        SWE::Global::g * (0.5 * pow(internal.ze_at_gp[gp], 2) + internal.ze_at_gp[gp] *
                internal.bath_at_gp[gp]);
        internal.qx_flux_at_gp[GlobalCoord::y][gp] = internal.qx_at_gp[gp] * internal.qy_at_gp[gp] /
                internal.water_column_hgt_at_gp[gp];
        internal.qy_flux_at_gp[GlobalCoord::x][gp] = internal.qx_at_gp[gp] * internal.qy_at_gp[gp] /
                internal.water_column_hgt_at_gp[gp];
        internal.qy_flux_at_gp[GlobalCoord::y][gp] = pow(internal.qy_at_gp[gp], 2) /
                internal.water_column_hgt_at_gp[gp] + SWE::Global::g * (0.5 *
                pow(internal.ze_at_gp[gp], 2) + internal.ze_at_gp[gp] * internal.bath_at_gp[gp]);
    }
}

template <typename ElementType>
__global__ void cuda_volume_kernel2(ElementType& elt, uint stage, uint n_dof) {
    auto& internal = elt.data.internal;
    auto& state = elt.data.state[stage];
    uint dof = blockDim.x * blockIdx.x + threadIdx.x;
    if (dof < n_dof && dof > 0) {
        // skip dof = 0, which is a constant and thus trivially 0 NOT ALWAYS!
        state.rhs_ze[dof] = elt.IntegrationDPhi(GlobalCoord::x, dof,
                internal.ze_flux_at_gp[GlobalCoord::x]) + elt.IntegrationDPhi(GlobalCoord::y, dof,
                internal.ze_flux_at_gp[GlobalCoord::y]);
        state.rhs_qx[dof] = elt.IntegrationDPhi(GlobalCoord::x, dof,
                internal.qx_flux_at_gp[GlobalCoord::x]) + elt.IntegrationDPhi(GlobalCoord::y, dof,
                internal.qx_flux_at_gp[GlobalCoord::y]);
        state.rhs_qy[dof] = elt.IntegrationDPhi(GlobalCoord::x, dof,
                internal.qy_flux_at_gp[GlobalCoord::x]) + elt.IntegrationDPhi(GlobalCoord::y, dof,
                internal.qy_flux_at_gp[GlobalCoord::y]);
    }
}

template <typename ElementType>
__global__ void cuda_source_kernel1(ElementType& elt, uint n_gp) {
    auto& internal = elt.data.internal;
    uint gp = blockDim.x * blockIdx.x + threadIdx.x;
    if (gp < n_gp) {
            // compute contribution of hydrostatic pressure
            internal.qx_source_term_at_gp[gp] += SWE::Global::g * internal.bath_deriv_wrt_x_at_gp[gp] *
                    internal.ze_at_gp[gp];
            internal.qy_source_term_at_gp[gp] += SWE::Global::g * internal.bath_deriv_wrt_y_at_gp[gp] *
                    internal.ze_at_gp[gp];
            double u_at_gp = internal.qx_at_gp[gp] / internal.water_column_hgt_at_gp[gp];
            double v_at_gp = internal.qy_at_gp[gp] / internal.water_column_hgt_at_gp[gp];

            // compute bottom friction contribution
            double bottom_friction_stress = SWE::Global::Cf * std::hypot(u_at_gp, v_at_gp) /
                    internal.water_column_hgt_at_gp[gp];
            internal.qx_source_term_at_gp[gp] -= bottom_friction_stress * internal.qx_at_gp[gp];
            internal.qy_source_term_at_gp[gp] -= bottom_friction_stress * internal.qy_at_gp[gp];
    }
}

template <typename ElementType>
__global__ void cuda_source_kernel2(ElementType& elt, uint stage, uint n_dof) {
    auto& internal = elt.data.internal;
    auto& state = elt.data.state[stage];
    uint dof = blockDim.x * blockIdx.x + threadIdx.x;
    if (dof < n_dof) {
        state.rhs_ze[dof] += elt.IntegrationPhi(dof, internal.ze_source_term_at_gp);
        state.rhs_qx[dof] += elt.IntegrationPhi(dof, internal.qx_source_term_at_gp);
        state.rhs_qy[dof] += elt.IntegrationPhi(dof, internal.qy_source_term_at_gp);
    }
}

template <typename InterfaceType>
__global__ void cuda_interface_kernel1(InterfaceType& interface, uint n_gp) {
  uint gp = blockDim.x * blockIdx.x + threadIdx.x;
  auto& boundary_ex = interface.data_ex.boundary[interface.bound_id_ex];
  auto& boundary_in = interface.data_in.boundary[interface.bound_id_in];
  if (gp < n_gp) {
        uint gp_ex = interface.data_in.get_ngp_boundary(interface.bound_id_in) - gp - 1;

        LLF_flux(boundary_in.ze_at_gp[gp],
                 boundary_ex.ze_at_gp[gp_ex],
                 boundary_in.qx_at_gp[gp],
                 boundary_ex.qx_at_gp[gp_ex],
                 boundary_in.qy_at_gp[gp],
                 boundary_ex.qy_at_gp[gp_ex],
                 boundary_in.bath_at_gp[gp],
                 interface.surface_normal[gp],
                 boundary_in.ze_numerical_flux_at_gp[gp],
                 boundary_in.qx_numerical_flux_at_gp[gp],
                 boundary_in.qy_numerical_flux_at_gp[gp]);

        boundary_ex.ze_numerical_flux_at_gp[gp_ex] = -boundary_in.ze_numerical_flux_at_gp[gp];
        boundary_ex.qx_numerical_flux_at_gp[gp_ex] = -boundary_in.qx_numerical_flux_at_gp[gp];
        boundary_ex.qy_numerical_flux_at_gp[gp_ex] = -boundary_in.qy_numerical_flux_at_gp[gp];
    }
}

template <typename InterfaceType>
__global__ void cuda_interface_kernel2(InterfaceType& interface, uint stage, uint n_dof) {
    uint dof = blockIdx.x * blockDim.x + threadIdx.x;
    auto& state_in = interface.data_in.state[stage];
    auto& boundary_in = interface.data_in.boundary[interface.bound_id_in];
    if (dof < n_dof) {
        state_in.rhs_ze[dof] -= interface.IntegrationPhiIN(dof, boundary_in.ze_numerical_flux_at_gp);
        state_in.rhs_qx[dof] -= interface.IntegrationPhiIN(dof, boundary_in.qx_numerical_flux_at_gp);
        state_in.rhs_qy[dof] -= interface.IntegrationPhiIN(dof, boundary_in.qy_numerical_flux_at_gp);
    }
}

template <typename InterfaceType>
__global__ void cuda_interface_kernel3(InterfaceType& interface, uint stage, uint n_dof) {
    uint dof = blockIdx.x * blockDim.x + threadIdx.x;
    auto& state_ex = interface.data_ex.state[stage];
    auto& boundary_ex = interface.data_ex.boundary[interface.bound_id_ex];
    if (dof < n_dof) {
        state_ex.rhs_ze[dof] -= interface.IntegrationPhiEX(dof, boundary_ex.ze_numerical_flux_at_gp);
        state_ex.rhs_qx[dof] -= interface.IntegrationPhiEX(dof, boundary_ex.qx_numerical_flux_at_gp);
        state_ex.rhs_qy[dof] -= interface.IntegrationPhiEX(dof, boundary_ex.qy_numerical_flux_at_gp);
    }
}

template <typename BoundaryType>
__global__ void cuda_boundary_kernel1(const Stepper& stepper, BoundaryType& bound, uint n_gp) {
    uint gp = blockIdx.x * blockDim.x + threadIdx.x;
    auto& boundary = bound.data.boundary[bound.bound_id];
    if (gp < n_gp) {
        double ze_ex, qx_ex, qy_ex;
        bound.boundary_condition.GetEX(stepper, gp, bound.surface_normal, boundary.ze_at_gp,
                boundary.qx_at_gp, boundary.qy_at_gp, ze_ex, qx_ex, qy_ex);
        LLF_flux(boundary.ze_at_gp[gp], ze_ex, boundary.qx_at_gp[gp], qx_ex, boundary.qy_at_gp[gp],
                qy_ex, boundary.bath_at_gp[gp], bound.surface_normal[gp],
                boundary.ze_numerical_flux_at_gp[gp], boundary.qx_numerical_flux_at_gp[gp],
                boundary.qy_numerical_flux_at_gp[gp]);
    }
}

template <typename BoundaryType>
__global__ void cuda_boundary_kernel2(const Stepper& stepper, BoundaryType& bound, uint n_dof) {
    uint dof = blockIdx.x * blockDim.x + threadIdx.x;
    auto& boundary = bound.data.boundary[bound.bound_id];
    const uint stage = stepper.get_stage();
    auto& state = bound.data.state[stage];
    if (dof < n_dof) {
        state.rhs_ze[dof] -= bound.IntegrationPhi(dof, boundary.ze_numerical_flux_at_gp);
        state.rhs_qx[dof] -= bound.IntegrationPhi(dof, boundary.qx_numerical_flux_at_gp);
        state.rhs_qy[dof] -= bound.IntegrationPhi(dof, boundary.qy_numerical_flux_at_gp);
    }
}

template <typename ElementType>
__global__ void cuda_update_kernel(const Stepper& stepper, ElementType& elt, uint stage,
        uint n_dof) {
    uint dof = blockIdx.x * blockDim.x + threadIdx.x;
    auto& state = elt.data.state;
    auto& curr_state = elt.data.state[stage];
    auto& next_state = elt.data.state[stage + 1];
    double dt = stepper.get_dt();
    if (dof < n_dof) {
        for (uint s = 0; s <= stage; ++s) {
            next_state.ze[dof] += stepper.ark[stage][s] * state[s].ze[dof] + dt *
                    stepper.brk[stage][s] * state[s].rhs_ze[dof];
            next_state.qx[dof] += stepper.ark[stage][s] * state[s].qx[dof] + dt *
                    stepper.brk[stage][s] * state[s].rhs_qx[dof];
            next_state.qy[dof] += stepper.ark[stage][s] * state[s].qy[dof] + dt *
                    stepper.brk[stage][s] * state[s].rhs_qy[dof];
        }
    }
}

template <typename ElementType>
void SWE::CUDAProblem::volume_kernel(const Stepper& stepper, ElementType& elt) {
    const uint stage = stepper.get_stage();
    auto& state = elt.data.state[stage];
    auto& internal = elt.data.internal;

    // get state at Gauss points
    elt.ComputeUgp(state.ze, internal.ze_at_gp);
    elt.ComputeUgp(state.qx, internal.qx_at_gp);
    elt.ComputeUgp(state.qy, internal.qy_at_gp);

    // assemble flux
    uint blocksPerGrid = (elt.data.get_ngp_internal() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    cuda_volume_kernel1<ElementType><<<blocksPerGrid, THREADS_PER_BLOCK>>>(elt,
            elt.data.get_ngp_internal());
    blocksPerGrid = (elt.data.get_ndof() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    cuda_volume_kernel2<ElementType><<<blocksPerGrid, THREADS_PER_BLOCK>>>(elt, stage,
            elt.data.get_ndof());
}

template <typename ElementType>
void SWE::CUDAProblem::source_kernel(const Stepper& stepper, ElementType& elt) {
    const uint stage = stepper.get_stage();
    auto& state = elt.data.state[stage];
    auto& internal = elt.data.internal;
    double t = stepper.get_t_at_curr_stage();
    auto source_ze = [t](Point<2>& pt) { return SWE::source_ze(t, pt); };
    auto source_qx = [t](Point<2>& pt) { return SWE::source_qx(t, pt); };
    auto source_qy = [t](Point<2>& pt) { return SWE::source_qy(t, pt); };

    elt.ComputeFgp(source_ze, internal.ze_source_term_at_gp);
    elt.ComputeFgp(source_qx, internal.qx_source_term_at_gp);
    elt.ComputeFgp(source_qy, internal.qy_source_term_at_gp);

    // note we assume that the values at gauss points have already been computed

    uint blocksPerGrid = (elt.data.get_ngp_internal() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    cuda_source_kernel1<ElementType><<<blocksPerGrid, THREADS_PER_BLOCK>>>(elt,
            elt.data.get_ngp_internal());
    blocksPerGrid = (elt.data.get_ndof() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    cuda_source_kernel2<ElementType><<<blocksPerGrid, THREADS_PER_BLOCK>>>(elt, stage,
            elt.data.get_ndof());
}

template <typename InterfaceType>
void SWE::CUDAProblem::interface_kernel(const Stepper& stepper, InterfaceType& interface) {
    const uint stage = stepper.get_stage();
    auto& state_in = interface.data_in.state[stage];
    auto& state_ex = interface.data_ex.state[stage];
    auto& boundary_in = interface.data_in.boundary[interface.bound_id_in];
    auto& boundary_ex = interface.data_ex.boundary[interface.bound_id_ex];
    interface.ComputeUgpIN(state_in.ze, boundary_in.ze_at_gp);
    interface.ComputeUgpIN(state_in.qx, boundary_in.qx_at_gp);
    interface.ComputeUgpIN(state_in.qy, boundary_in.qy_at_gp);

    interface.ComputeUgpEX(state_ex.ze, boundary_ex.ze_at_gp);
    interface.ComputeUgpEX(state_ex.qx, boundary_ex.qx_at_gp);
    interface.ComputeUgpEX(state_ex.qy, boundary_ex.qy_at_gp);

    // assemble numerical fluxes
    uint blocksPerGrid = (interface.data_in.get_ngp_boundary(interface.bound_id_in) +
            THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    cuda_interface_kernel1<InterfaceType><<<blocksPerGrid, THREADS_PER_BLOCK>>>(interface,
            interface.data_in.get_ngp_boundary(interface.bound_id_in));

    // now compute contributions to the righthand side
    blocksPerGrid = (interface.data_in.get_ndof() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    cuda_interface_kernel2<InterfaceType><<<blocksPerGrid, THREADS_PER_BLOCK>>>(interface, stage,
            interface.data_in.get_ndof());
    blocksPerGrid = (interface.data_ex.get_ndof() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    cuda_interface_kernel3<InterfaceType><<<blocksPerGrid, THREADS_PER_BLOCK>>>(interface, stage,
            interface.data_ex.get_ndof());
}

template <typename BoundaryType>
void SWE::CUDAProblem::boundary_kernel(const Stepper& stepper, BoundaryType& bound) {
    const uint stage = stepper.get_stage();

    auto& state = bound.data.state[stage];
    auto& boundary = bound.data.boundary[bound.bound_id];

    bound.ComputeUgp(state.ze, boundary.ze_at_gp);
    bound.ComputeUgp(state.qx, boundary.qx_at_gp);
    bound.ComputeUgp(state.qy, boundary.qy_at_gp);

    uint blocksPerGrid = (bound.data.get_ngp_boundary(bound.bound_id) + THREADS_PER_BLOCK - 1) /
            THREADS_PER_BLOCK;
    cuda_boundary_kernel1<BoundaryType><<<blocksPerGrid, THREADS_PER_BLOCK>>>(stepper, bound,
            bound.data.get_ngp_boundary(bound.bound_id));

    // now compute contributions to the righthand side
    blocksPerGrid = (bound.data.get_ndof() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    cuda_boundary_kernel2<BoundaryType><<<blocksPerGrid, THREADS_PER_BLOCK>>>(stepper, bound,
            bound.data.get_ndof());
}

template <typename ElementType>
void SWE::CUDAProblem::update_kernel(const Stepper& stepper, ElementType& elt) {
    const uint stage = stepper.get_stage();

    auto& state = elt.data.state;
    auto& curr_state = elt.data.state[stage];
    auto& next_state = elt.data.state[stage + 1];

    curr_state.rhs_ze = elt.SolveLSE(curr_state.rhs_ze);
    curr_state.rhs_qx = elt.SolveLSE(curr_state.rhs_qx);
    curr_state.rhs_qy = elt.SolveLSE(curr_state.rhs_qy);

    std::fill(next_state.ze.begin(), next_state.ze.end(), 0);
    std::fill(next_state.qx.begin(), next_state.qx.end(), 0);
    std::fill(next_state.qy.begin(), next_state.qy.end(), 0);

    uint blocksPerGrid = (elt.data.get_ndof() + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    cuda_update_kernel<ElementType><<<blocksPerGrid, THREADS_PER_BLOCK>>>(stepper, elt, stage,
            elt.data.get_ndof());
}

template <typename ElementType>
void SWE::CUDAProblem::swap_states_kernel(const Stepper& stepper, ElementType& elt) {
    uint n_stages = stepper.get_num_stages();
    auto& state = elt.data.state;

    std::swap(state[0].ze, state[n_stages].ze);
    std::swap(state[0].qx, state[n_stages].qx);
    std::swap(state[0].qy, state[n_stages].qy);
}

template <typename ElementType>
void SWE::CUDAProblem::scrutinize_solution_kernel(const Stepper& stepper, ElementType& elt) {
    uint stage = stepper.get_stage();

    auto& state = elt.data.state[stage];

    for (auto& ze_mode : state.ze) {
        if (std::isnan(ze_mode)) {
            std::cerr << "Error: found isnan ze at Element " << elt.GetID();
            std::cerr << "       At stage: " << stage << "\n";
        }
    }

    for (auto& qx_mode : state.qx) {
        if (std::isnan(qx_mode)) {
            std::cerr << "Error: found isnan qx at Element " << elt.GetID();
            std::cerr << "       At stage: " << stage << "\n";
        }
    }

    for (auto& qy_mode : state.qy) {
        if (std::isnan(qy_mode)) {
            std::cerr << "Error: found isnan qy at Element " << elt.GetID();
            std::cerr << "       At stage: " << stage << "\n";
        }
    }

    for (auto& rhs_ze_mode : state.rhs_ze) {
        if (std::isnan(rhs_ze_mode)) {
            std::cerr << "Error: found isnan rhs_ze at Element " << elt.GetID();
            std::cerr << "       At stage: " << stage << "\n";
        }
    }

    for (auto& rhs_qx_mode : state.rhs_qx) {
        if (std::isnan(rhs_qx_mode)) {
            std::cerr << "Error: found isnan rhs_qx at Element " << elt.GetID();
            std::cerr << "       At stage: " << stage << "\n";
        }
    }

    for (auto& rhs_qy_mode : state.rhs_qy) {
        if (std::isnan(rhs_qy_mode)) {
            std::cerr << "Error: found isnan rhs_qy at Element " << elt.GetID();
            std::cerr << "       At stage: " << stage << "\n";
        }
    }
}
