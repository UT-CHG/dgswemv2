#include "hip/hip_runtime.h"
#include "swe_cuda_kernels.cuh"
#include "swe_definitions.hpp"
#include "general_definitions.hpp"

template <typename ElementType>
void cuda_volume_kernel1(uint block_dim, uint thread_dim, ElementType& elt, uint n_gp) {
    std::cout << "running" << std::endl;
    cuda_volume_kernel1<ElementType><<<block_dim, thread_dim>>>(elt, n_gp);
}

template <typename ElementType>
__global__ void cuda_volume_kernel1(ElementType& elt, uint n_gp) {
    auto& internal = elt.data.internal;
    uint gp = blockDim.x * blockIdx.x + threadIdx.x;
    if (gp < n_gp) {
        internal.water_column_hgt_at_gp[gp] = internal.ze_at_gp[gp] + internal.bath_at_gp[gp];
        internal.ze_flux_at_gp[GlobalCoord::x][gp] = internal.qx_at_gp[gp];
        internal.ze_flux_at_gp[GlobalCoord::y][gp] = internal.qy_at_gp[gp];
        internal.qx_flux_at_gp[GlobalCoord::x][gp] = pow(internal.qx_at_gp[gp], 2) /
                internal.water_column_hgt_at_gp[gp] +
        SWE::Global::g * (0.5 * pow(internal.ze_at_gp[gp], 2) + internal.ze_at_gp[gp] *
                internal.bath_at_gp[gp]);
        internal.qx_flux_at_gp[GlobalCoord::y][gp] = internal.qx_at_gp[gp] * internal.qy_at_gp[gp] /
                internal.water_column_hgt_at_gp[gp];
        internal.qy_flux_at_gp[GlobalCoord::x][gp] = internal.qx_at_gp[gp] * internal.qy_at_gp[gp] /
                internal.water_column_hgt_at_gp[gp];
        internal.qy_flux_at_gp[GlobalCoord::y][gp] = pow(internal.qy_at_gp[gp], 2) /
                internal.water_column_hgt_at_gp[gp] + SWE::Global::g * (0.5 *
                pow(internal.ze_at_gp[gp], 2) + internal.ze_at_gp[gp] * internal.bath_at_gp[gp]);
    }
}

template <typename ElementType>
void cuda_volume_kernel2(uint block_dim, uint thread_dim, ElementType& elt, uint stage,
        uint n_dof) {
    std::cout << "running" << std::endl;
    cuda_volume_kernel2<ElementType><<<block_dim, thread_dim>>>(elt, stage, n_dof);
}

template <typename ElementType>
__global__ void cuda_volume_kernel2(ElementType& elt, uint stage, uint n_dof) {
    auto& internal = elt.data.internal;
    auto& state = elt.data.state[stage];
    uint dof = blockDim.x * blockIdx.x + threadIdx.x;
    if (dof < n_dof && dof > 0) {
        // skip dof = 0, which is a constant and thus trivially 0 NOT ALWAYS!
        state.rhs_ze[dof] = elt.IntegrationDPhi(GlobalCoord::x, dof,
                internal.ze_flux_at_gp[GlobalCoord::x]) + elt.IntegrationDPhi(GlobalCoord::y, dof,
                internal.ze_flux_at_gp[GlobalCoord::y]);
        state.rhs_qx[dof] = elt.IntegrationDPhi(GlobalCoord::x, dof,
                internal.qx_flux_at_gp[GlobalCoord::x]) + elt.IntegrationDPhi(GlobalCoord::y, dof,
                internal.qx_flux_at_gp[GlobalCoord::y]);
        state.rhs_qy[dof] = elt.IntegrationDPhi(GlobalCoord::x, dof,
                internal.qy_flux_at_gp[GlobalCoord::x]) + elt.IntegrationDPhi(GlobalCoord::y, dof,
                internal.qy_flux_at_gp[GlobalCoord::y]);
    }
}

template <typename ElementType>
void cuda_source_kernel1(uint block_dim, uint thread_dim, ElementType& elt, uint n_gp) {
    std::cout << "running" << std::endl;
    cuda_source_kernel1<ElementType><<<block_dim, thread_dim>>>(elt, n_gp);
}

template <typename ElementType>
__global__ void cuda_source_kernel1(ElementType& elt, uint n_gp) {
    auto& internal = elt.data.internal;
    uint gp = blockDim.x * blockIdx.x + threadIdx.x;
    if (gp < n_gp) {
            // compute contribution of hydrostatic pressure
            internal.qx_source_term_at_gp[gp] += SWE::Global::g * internal.bath_deriv_wrt_x_at_gp[gp] *
                    internal.ze_at_gp[gp];
            internal.qy_source_term_at_gp[gp] += SWE::Global::g * internal.bath_deriv_wrt_y_at_gp[gp] *
                    internal.ze_at_gp[gp];
            double u_at_gp = internal.qx_at_gp[gp] / internal.water_column_hgt_at_gp[gp];
            double v_at_gp = internal.qy_at_gp[gp] / internal.water_column_hgt_at_gp[gp];

            // compute bottom friction contribution
            double bottom_friction_stress = SWE::Global::Cf * std::hypot(u_at_gp, v_at_gp) /
                    internal.water_column_hgt_at_gp[gp];
            internal.qx_source_term_at_gp[gp] -= bottom_friction_stress * internal.qx_at_gp[gp];
            internal.qy_source_term_at_gp[gp] -= bottom_friction_stress * internal.qy_at_gp[gp];
    }
}

template <typename ElementType>
void cuda_source_kernel2(uint block_dim, uint thread_dim, ElementType& elt, uint stage,
        uint n_dof) {
    std::cout << "running" << std::endl;
    cuda_source_kernel2<ElementType><<<block_dim, thread_dim>>>(elt, stage, n_dof);
}

template <typename ElementType>
__global__ void cuda_source_kernel2(ElementType& elt, uint stage, uint n_dof) {
    auto& internal = elt.data.internal;
    auto& state = elt.data.state[stage];
    uint dof = blockDim.x * blockIdx.x + threadIdx.x;
    if (dof < n_dof) {
        state.rhs_ze[dof] += elt.IntegrationPhi(dof, internal.ze_source_term_at_gp);
        state.rhs_qx[dof] += elt.IntegrationPhi(dof, internal.qx_source_term_at_gp);
        state.rhs_qy[dof] += elt.IntegrationPhi(dof, internal.qy_source_term_at_gp);
    }
}

template <typename InterfaceType>
void cuda_interface_kernel1(uint block_dim, uint thread_dim, InterfaceType& interface, uint n_gp) {
    std::cout << "running" << std::endl;
    cuda_interface_kernel1<InterfaceType><<<block_dim, thread_dim>>>(interface, n_gp);
}

template <typename InterfaceType>
__global__ void cuda_interface_kernel1(InterfaceType& interface, uint n_gp) {
  uint gp = blockDim.x * blockIdx.x + threadIdx.x;
  auto& boundary_ex = interface.data_ex.boundary[interface.bound_id_ex];
  auto& boundary_in = interface.data_in.boundary[interface.bound_id_in];
  if (gp < n_gp) {
        uint gp_ex = interface.data_in.get_ngp_boundary(interface.bound_id_in) - gp - 1;

        LLF_flux(boundary_in.ze_at_gp[gp],
                 boundary_ex.ze_at_gp[gp_ex],
                 boundary_in.qx_at_gp[gp],
                 boundary_ex.qx_at_gp[gp_ex],
                 boundary_in.qy_at_gp[gp],
                 boundary_ex.qy_at_gp[gp_ex],
                 boundary_in.bath_at_gp[gp],
                 interface.surface_normal[gp],
                 boundary_in.ze_numerical_flux_at_gp[gp],
                 boundary_in.qx_numerical_flux_at_gp[gp],
                 boundary_in.qy_numerical_flux_at_gp[gp]);

        boundary_ex.ze_numerical_flux_at_gp[gp_ex] = -boundary_in.ze_numerical_flux_at_gp[gp];
        boundary_ex.qx_numerical_flux_at_gp[gp_ex] = -boundary_in.qx_numerical_flux_at_gp[gp];
        boundary_ex.qy_numerical_flux_at_gp[gp_ex] = -boundary_in.qy_numerical_flux_at_gp[gp];
    }
}

template <typename InterfaceType>
void cuda_interface_kernel2(uint block_dim, uint thread_dim, InterfaceType& interface, uint stage,
       uint n_dof) {
    std::cout << "running" << std::endl;
    cuda_interface_kernel2<InterfaceType><<<block_dim, thread_dim>>>(interface, stage, n_dof);
}

template <typename InterfaceType>
__global__ void cuda_interface_kernel2(InterfaceType& interface, uint stage, uint n_dof) {
    uint dof = blockIdx.x * blockDim.x + threadIdx.x;
    auto& state_in = interface.data_in.state[stage];
    auto& boundary_in = interface.data_in.boundary[interface.bound_id_in];
    if (dof < n_dof) {
        state_in.rhs_ze[dof] -= interface.IntegrationPhiIN(dof, boundary_in.ze_numerical_flux_at_gp);
        state_in.rhs_qx[dof] -= interface.IntegrationPhiIN(dof, boundary_in.qx_numerical_flux_at_gp);
        state_in.rhs_qy[dof] -= interface.IntegrationPhiIN(dof, boundary_in.qy_numerical_flux_at_gp);
    }
}

template <typename InterfaceType>
void cuda_interface_kernel3(uint block_dim, uint thread_dim, InterfaceType& interface, uint stage,
        uint n_dof) {
    std::cout << "running" << std::endl;
  cuda_interface_kernel3<InterfaceType><<<block_dim, thread_dim>>>(interface, stage, n_dof);
}

template <typename InterfaceType>
__global__ void cuda_interface_kernel3(InterfaceType& interface, uint stage, uint n_dof) {
    uint dof = blockIdx.x * blockDim.x + threadIdx.x;
    auto& state_ex = interface.data_ex.state[stage];
    auto& boundary_ex = interface.data_ex.boundary[interface.bound_id_ex];
    if (dof < n_dof) {
        state_ex.rhs_ze[dof] -= interface.IntegrationPhiEX(dof, boundary_ex.ze_numerical_flux_at_gp);
        state_ex.rhs_qx[dof] -= interface.IntegrationPhiEX(dof, boundary_ex.qx_numerical_flux_at_gp);
        state_ex.rhs_qy[dof] -= interface.IntegrationPhiEX(dof, boundary_ex.qy_numerical_flux_at_gp);
    }
}

template <typename BoundaryType>
void cuda_boundary_kernel1(uint block_dim, uint thread_dim, const Stepper& stepper,
        BoundaryType& bound, uint n_gp) {
    std::cout << "running" << std::endl;
    cuda_boundary_kernel1<BoundaryType><<<block_dim, thread_dim>>>(stepper, bound, n_gp);
}

template <typename BoundaryType>
__global__ void cuda_boundary_kernel1(const Stepper& stepper, BoundaryType& bound, uint n_gp) {
    uint gp = blockIdx.x * blockDim.x + threadIdx.x;
    auto& boundary = bound.data.boundary[bound.bound_id];
    if (gp < n_gp) {
        double ze_ex, qx_ex, qy_ex;
        bound.boundary_condition.GetEX(stepper, gp, bound.surface_normal, boundary.ze_at_gp,
                boundary.qx_at_gp, boundary.qy_at_gp, ze_ex, qx_ex, qy_ex);
        LLF_flux(boundary.ze_at_gp[gp], ze_ex, boundary.qx_at_gp[gp], qx_ex, boundary.qy_at_gp[gp],
                qy_ex, boundary.bath_at_gp[gp], bound.surface_normal[gp],
                boundary.ze_numerical_flux_at_gp[gp], boundary.qx_numerical_flux_at_gp[gp],
                boundary.qy_numerical_flux_at_gp[gp]);
    }
}

template <typename BoundaryType>
void cuda_boundary_kernel2(uint block_dim, uint thread_dim, const Stepper& stepper,
        BoundaryType& bound, uint n_dof) {
    std::cout << "running" << std::endl;
    cuda_boundary_kernel2<BoundaryType><<<block_dim, thread_dim>>>(stepper, bound, n_dof);
}

template <typename BoundaryType>
__global__ void cuda_boundary_kernel2(const Stepper& stepper, BoundaryType& bound, uint n_dof) {
    uint dof = blockIdx.x * blockDim.x + threadIdx.x;
    auto& boundary = bound.data.boundary[bound.bound_id];
    const uint stage = stepper.get_stage();
    auto& state = bound.data.state[stage];
    if (dof < n_dof) {
        state.rhs_ze[dof] -= bound.IntegrationPhi(dof, boundary.ze_numerical_flux_at_gp);
        state.rhs_qx[dof] -= bound.IntegrationPhi(dof, boundary.qx_numerical_flux_at_gp);
        state.rhs_qy[dof] -= bound.IntegrationPhi(dof, boundary.qy_numerical_flux_at_gp);
    }
}

template <typename ElementType>
void cuda_update_kernel(uint block_dim, uint thread_dim, const Stepper& stepper, ElementType& elt,
        uint stage, uint n_dof) {
    std::cout << "running" << std::endl;
    cuda_update_kernel<ElementType><<<block_dim, thread_dim>>>(stepper, elt, stage, n_dof);
}

template <typename ElementType>
__global__ void cuda_update_kernel(const Stepper& stepper, ElementType& elt, uint stage,
        uint n_dof) {
    uint dof = blockIdx.x * blockDim.x + threadIdx.x;
    auto& state = elt.data.state;
    auto& curr_state = elt.data.state[stage];
    auto& next_state = elt.data.state[stage + 1];
    double dt = stepper.get_dt();
    if (dof < n_dof) {
        for (uint s = 0; s <= stage; ++s) {
            next_state.ze[dof] += stepper.ark[stage][s] * state[s].ze[dof] + dt *
                    stepper.brk[stage][s] * state[s].rhs_ze[dof];
            next_state.qx[dof] += stepper.ark[stage][s] * state[s].qx[dof] + dt *
                    stepper.brk[stage][s] * state[s].rhs_qx[dof];
            next_state.qy[dof] += stepper.ark[stage][s] * state[s].qy[dof] + dt *
                    stepper.brk[stage][s] * state[s].rhs_qy[dof];
        }
    }
}

